﻿#define _USE_MATH_DEFINES

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>


static uint8_t* buffer=0;
static double* vecl=0;

__global__ void addKernel(uint8_t* buffer, double* vecl, bool flipg, double dc, double uhpy, double pos0,double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, double bhsize, double whsize)
{
	double vecn0, vecn1, vecn2;
	double roomsize,schecker;


	bool flip;
	double sx, sy, sz;
	double dotp;

	double leangle,lecos,lesign,lecossq,lerayon,lex1,lex2,lexw;
	double vecperp0, vecperp1, vecperp2;
	double vecperpn;
	
	double exitlgt, exitangle, exitangle2;

	double pos20, pos21, pos22;
	double vecn20, vecn21, vecn22;
	double qa, qb, qc;
	double discr;
	double tcont;
	double tmpr;
	double tmpx2;

	double tmin,tsol;
	int tmincoord;
	double tmpsign;

	double coll0, coll1;
	int checker;
	int ctmp0, ctmp1, ctmp2;
	int dtmp0, dtmp1, dtmp2;

	int tmp = blockIdx.x * blockDim.x + threadIdx.x;
	int tmpx = tmp % 1280;
	int tmpy = (tmp-tmpx) /1280;
	flip = false;

	/////

	roomsize = 10;
	schecker = 1;
	/////

	vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
	vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
	vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

	vecn0 /= vecl[tmp];
	vecn1 /= vecl[tmp];
	vecn2 /= vecl[tmp];

	if (dc < 1)
	{
		sx = pos0/dc;
		sy = pos1/dc;
		sz = pos2/dc;

		dotp = sx * vecn0 + sy * vecn1 + sz * vecn2;
		leangle = acos(dotp);

		vecperp0 = vecn0 - dotp * sx;
		vecperp1 = vecn1 - dotp * sy;
		vecperp2 = vecn2 - dotp * sz;

		vecperpn = sqrt(vecperp0 * vecperp0 + vecperp1 * vecperp1 + vecperp2 * vecperp2);

		vecperp0 /= vecperpn;
		vecperp1 /= vecperpn;
		vecperp2 /= vecperpn;

		lecos = cos(leangle);
		if (lecos < 0) lesign = -1;
		else lesign = 1;

		lecossq = lecos * lecos;


		lerayon = uhpy * sqrt(1 / (1 - lecossq));


		lex1 = sqrt(lerayon * lerayon - uhpy * uhpy) * lesign;
		lex2 = sqrt(lerayon * lerayon - 1);



		if (lesign<0 && lerayon>whsize)
		{
			flip = true;

			lexw = sqrt(lerayon * lerayon - whsize * whsize);

			exitlgt = -lex1 + lex2 - 2 * lexw;
			exitangle = fmod(exitlgt,bhsize);
			exitangle = (2 * M_PI / bhsize) * exitangle;

			exitangle2 = acos((1 / uhpy) * sqrt(uhpy * uhpy - 1 + lecossq));


			pos20 = cos(exitangle) * sx + sin(exitangle) * vecperp0;
			pos21 = cos(exitangle) * sy + sin(exitangle) * vecperp1;
			pos22 = cos(exitangle) * sz + sin(exitangle) * vecperp2;

			vecn20 = cos(exitangle + exitangle2) * sx + sin(exitangle2 + exitangle) * vecperp0;
			vecn21 = cos(exitangle + exitangle2) * sy + sin(exitangle2 + exitangle) * vecperp1;
			vecn22 = cos(exitangle + exitangle2) * sz + sin(exitangle2 + exitangle) * vecperp2;
		}
		else
		{
			exitlgt = lex2 - lex1;
			exitangle = fmod(exitlgt,bhsize);
			exitangle = (2 * M_PI / bhsize) * exitangle;

			exitangle2 = acos((1 / uhpy) * sqrt(uhpy * uhpy - 1 + lecossq));

			pos20 = cos(exitangle) * sx + sin(exitangle) * vecperp0;
			pos21 = cos(exitangle) * sy + sin(exitangle) * vecperp1;
			pos22 = cos(exitangle) * sz + sin(exitangle) * vecperp2;


			vecn20 = cos(exitangle + exitangle2) * sx + sin(exitangle2 + exitangle) * vecperp0;
			vecn21 = cos(exitangle + exitangle2) * sy + sin(exitangle2 + exitangle) * vecperp1;
			vecn22 = cos(exitangle + exitangle2) * sz + sin(exitangle2 + exitangle) * vecperp2;
		}
		tcont = 0;


	}
	else
	{

		qa = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
		qb = 2 * (vecn0 * pos0 + vecn1 * pos1 + vecn2 * pos2);
		qc = pos0 * pos0 + pos1 * pos1 + pos2 * pos2 - 1;

		discr = qb * qb - 4 * qa * qc;

		if (discr <= 0) {
			pos20 = pos0;
			pos21 = pos1;
			pos22 = pos2;

			vecn20 = vecn0;
			vecn21 = vecn1;
			vecn22 = vecn2;

			tcont = 0;
		}
		else
		{

			tcont = ((-1) * qb - sqrt(discr)) / (2 * qa);

			if (tcont > 0) {
				sx = vecn0 * tcont + pos0;
				sy = vecn1 * tcont + pos1;
				sz = vecn2 * tcont + pos2;

				dotp = sx * vecn0 + sy * vecn1 + sz * vecn2;
				leangle = acos(dotp);

				vecperp0 = vecn0 - dotp * sx;
				vecperp1 = vecn1 - dotp * sy;
				vecperp2 = vecn2 - dotp * sz;

				vecperpn = sqrt(vecperp0 * vecperp0 + vecperp1 * vecperp1 + vecperp2 * vecperp2);

				vecperp0 /= vecperpn;
				vecperp1 /= vecperpn;
				vecperp2 /= vecperpn;

				lecos = cos(leangle);
				lecossq = lecos * lecos;

				lerayon = sqrt(1 / (1 - lecossq));




				if (lerayon > whsize)
				{

					flip = true;


					lexw = sqrt((1 / (1 - lecossq)) - whsize * whsize);

					exitlgt = -2 * (lexw + lerayon * lecos);
					exitangle = fmod(exitlgt,bhsize);

					exitangle = (2 * M_PI / bhsize) * exitangle;

					pos20 = cos(exitangle) * sx + sin(exitangle) * vecperp0;
					pos21 = cos(exitangle) * sy + sin(exitangle) * vecperp1;
					pos22 = cos(exitangle) * sz + sin(exitangle) * vecperp2;


					vecn20 = cos(M_PI + exitangle - leangle) * sx + sin(M_PI - leangle + exitangle) * vecperp0;
					vecn21 = cos(M_PI + exitangle - leangle) * sy + sin(M_PI - leangle + exitangle) * vecperp1;
					vecn22 = cos(M_PI + exitangle - leangle) * sz + sin(M_PI - leangle + exitangle) * vecperp2;
				}
				else
				{
					exitlgt = -2 * lerayon * lecos;
					exitangle = fmod(exitlgt,bhsize);

					exitangle = (2 * M_PI / bhsize) * exitangle;


					pos20 = cos(exitangle) * sx + sin(exitangle) * vecperp0;
					pos21 = cos(exitangle) * sy + sin(exitangle) * vecperp1;
					pos22 = cos(exitangle) * sz + sin(exitangle) * vecperp2;


					vecn20 = cos(M_PI + exitangle - leangle) * sx + sin(M_PI - leangle + exitangle) * vecperp0;
					vecn21 = cos(M_PI + exitangle - leangle) * sy + sin(M_PI - leangle + exitangle) * vecperp1;
					vecn22 = cos(M_PI + exitangle - leangle) * sz + sin(M_PI - leangle + exitangle) * vecperp2;
				}
			}
			else {
				pos20 = pos0;
				pos21 = pos1;
				pos22 = pos2;

				vecn20 = vecn0;
				vecn21 = vecn1;
				vecn22 = vecn2;

				tcont = 0;
			}

		}
	}


	if (vecn20 < 0) tmpsign = -1;
	else tmpsign = 1;

	tmin = (tmpsign * roomsize - pos20) / vecn20;
	tmincoord = 0;

	if (vecn21 < 0) tmpsign = -1;
	else tmpsign = 1;
	tsol = (tmpsign * roomsize - pos21) / vecn21;
	if (tsol < tmin)
	{
		tmin = tsol;
		tmincoord = 1;
	}

	if (vecn22 < 0) tmpsign = -1;
	else tmpsign = 1;
	tsol = (tmpsign * roomsize - pos22) / vecn22;
	if (tsol < tmin)
	{
		tmin = tsol;
		tmincoord = 2;
	}

	if (tmincoord == 0)
	{
		coll0 = pos21 + tmin * vecn21;
		coll1 = pos22 + tmin * vecn22;
	}
	else if (tmincoord == 1)
	{
		coll0 = pos20 + tmin * vecn20;
		coll1 = pos22 + tmin * vecn22;
	}
	else
	{
		coll0 = pos20 + tmin * vecn20;
		coll1 = pos21 + tmin * vecn21;
	}

	checker = ((int)floor(coll0 * schecker)) % 2;
	checker += ((int)floor(coll1 * schecker)) % 2;
	if (checker < 0) checker += 2;
	checker %= 2;


	if (flip ^ flipg)
	{
		if (tmincoord == 0)
		{
			if (vecn20 < 0)
			{
				ctmp0 = 27;
				ctmp1 = 205;
				ctmp2 = 1;

				dtmp0 = 9;
				dtmp1 = 66;
				dtmp2 = 0;
			}
			else
			{
				ctmp0 = 232;
				ctmp1 = 220;
				ctmp2 = 0;

				dtmp0 = 85;
				dtmp1 = 81;
				dtmp2 = 0;
			}
		}
		else if (tmincoord == 1)
		{
			if (vecn21 < 0)
			{
				ctmp0 = 18;
				ctmp1 = 0;
				ctmp2 = 242;

				dtmp0 = 6;
				dtmp1 = 0;
				dtmp2 = 85;
			}
			else
			{
				ctmp0 = 234;
				ctmp1 = 0;
				ctmp2 = 6;

				dtmp0 = 89;
				dtmp1 = 0;
				dtmp2 = 2;
			}
		}
		else
		{
			if (vecn22 < 0)
			{
				ctmp0 = 4;
				ctmp1 = 255;
				ctmp2 = 255;

				dtmp0 = 0;
				dtmp1 = 81;
				dtmp2 = 81;
			}
			else
			{
				ctmp0 = 100;
				ctmp1 = 100;
				ctmp2 = 100;

				dtmp0 = 50;
				dtmp1 = 50;
				dtmp2 = 50;
			}
		}



		if (checker == 0) {
			buffer[4 * tmp] = dtmp0;
			buffer[4 * tmp + 1] = dtmp1;
			buffer[4 * tmp + 2] = dtmp2;
			buffer[4 * tmp + 3] = 255;
		}
		else
		{
			
			buffer[4 * tmp] = ctmp0;
			buffer[4 * tmp + 1] = ctmp1;
			buffer[4 * tmp + 2] = ctmp2;
			buffer[4 * tmp + 3] = 255;
		}
	}
	else
	{
		if (tmincoord == 2) {
			ctmp0 = 255;
			ctmp1 = 255;
			ctmp2 = 255;
		}
		else if (tmincoord == 0)
		{
			if (vecn20 < 0)
			{
				tmpx2 = (1.0/8.0) - (coll0 / (8.0 * roomsize));
				tmpr = fmod(tmpx2, 1.0/6.0);

				if (tmpx2 < 1.0/6.0)
				{
					ctmp0 = 255.0;
					ctmp1 = (int)(1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 1.0/3.0)
				{
					ctmp1 = 255.0;
					ctmp0 = (int)(255.0 - 1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 0.5)
				{
					ctmp1 = 255.0;
					ctmp2 = (int)(1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 2.0/3.0)
				{
					ctmp2 = 255.0;
					ctmp1 = (int)(255.0 - 1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 5.0 / 6.0)
				{
					ctmp2 = 255.0;
					ctmp0 = (int)(1530.0 * tmpr);
					ctmp1 = 0;
				}
				else
				{
					ctmp0 = 255.0;
					ctmp2 = (int)(255.0 - 1530.0 * tmpr);
					ctmp1 = 0;
				}
			}
			else
			{
				tmpx2 = (1.0 / 2.0) + (1.0 / 8.0) + coll0 / (8.0 * roomsize);

				tmpr = fmod(tmpx2, 1.0 / 6.0);

				if (tmpx2 < 1.0 / 6.0)
				{
					ctmp0 = 255.0;
					ctmp1 = (int)(1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 1.0 / 3.0)
				{
					ctmp1 = 255.0;
					ctmp0 = (int)(255.0 - 1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 0.5)
				{
					ctmp1 = 255.0;
					ctmp2 = (int)(1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 2.0 / 3.0)
				{
					ctmp2 = 255.0;
					ctmp1 = (int)(255.0 - 1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 5.0 / 6.0)
				{
					ctmp2 = 255.0;
					ctmp0 = (int)(1530.0 * tmpr);
					ctmp1 = 0;
				}
				else
				{
					ctmp0 = 255.0;
					ctmp2 = (int)(255.0 - 1530.0 * tmpr);
					ctmp1 = 0;
				}
			}
		}
		else
		{
			if (vecn21 < 0)
			{
				tmpx2 = (1.0 / 4.0) + (1.0 / 8.0) + coll0 / (8.0 * roomsize);
				tmpr = fmod(tmpx2, 1.0 / 6.0);

				if (tmpx2 < 1.0 / 6.0)
				{
					ctmp0 = 255.0;
					ctmp1 = (int)(1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 1.0 / 3.0)
				{
					ctmp1 = 255.0;
					ctmp0 = (int)(255.0 - 1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 0.5)
				{
					ctmp1 = 255.0;
					ctmp2 = (int)(1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 2.0 / 3.0)
				{
					ctmp2 = 255.0;
					ctmp1 = (int)(255.0 - 1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 5.0 / 6.0)
				{
					ctmp2 = 255.0;
					ctmp0 = (int)(1530.0 * tmpr);
					ctmp1 = 0;
				}
				else
				{
					ctmp0 = 255.0;
					ctmp2 = (int)(255.0 - 1530.0 * tmpr);
					ctmp1 = 0;
				}
			}
			else
			{
				tmpx2= (3.0 / 4.0) + (1.0 / 8.0) - coll0 / (8.0 * roomsize);

				tmpr = fmod(tmpx2, 1.0 / 6.0);

				if (tmpx2 < 1.0 / 6.0)
				{
					ctmp0 = 255.0;
					ctmp1 = (int)(1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 1.0 / 3.0)
				{
					ctmp1 = 255.0;
					ctmp0 = (int)(255.0 - 1530.0 * tmpr);
					ctmp2 = 0;
				}
				else if (tmpx2 < 0.5)
				{
					ctmp1 = 255.0;
					ctmp2 = (int)(1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 2.0 / 3.0)
				{
					ctmp2 = 255.0;
					ctmp1 = (int)(255.0 - 1530.0 * tmpr);
					ctmp0 = 0;
				}
				else if (tmpx2 < 5.0 / 6.0)
				{
					ctmp2 = 255.0;
					ctmp0 = (int)(1530.0 * tmpr);
					ctmp1 = 0;
				}
				else
				{
					ctmp0 = 255.0;
					ctmp2 = (int)(255.0 - 1530.0 * tmpr);
					ctmp1 = 0;
				}
			}
		}

		if (checker == 0) {
			buffer[4 * tmp] = 0;
			buffer[4 * tmp + 1] = 0;
			buffer[4 * tmp + 2] = 0;
			buffer[4 * tmp + 3] = 255;

		}
		else
		{
			buffer[4 * tmp] = ctmp0;
			buffer[4 * tmp + 1] = ctmp1;
			buffer[4 * tmp + 2] = ctmp2;
			buffer[4 * tmp + 3] = 255;
		}
		
	}
}

void cudaInit()
{
	double dist = 1;
	double sqsz = 0.01 / 4;
	int tmpx, tmpy;
	double* vecltmp = new double[1280 * 720];

	double vec0, vec1, vec2;
	double addy0, addy1, addy2;
	double addz0, addz1, addz2;
	double vecn0, vecn1, vecn2;
	double x00 = 1, x01 = 0, x02 = 0;
	double x10 = 0, x11 = 1, x12 = 0;
	double x20 = 0, x21 = 0, x22 = 1;
	double multy = (1 - 1280) * sqsz / 2;
	double multz = (720 - 1) * sqsz / 2;

	hipSetDevice(0);
	hipMalloc((void**)&buffer, 4 * 1280 * 720 * sizeof(uint8_t));
	hipMalloc((void**)&vecl, 1280 * 720 * sizeof(double));

	
	vec0 = dist * x00 + multy * x10 + multz * x20;
	vec1 = dist * x01 + multy * x11 + multz * x21;
	vec2 = dist * x02 + multy * x12 + multz * x22;
	
	addy0 = sqsz * x10;
	addy1 = sqsz * x11;
	addy2 = sqsz * x12;

	addz0 = -sqsz * x20;
	addz1 = -sqsz * x21;
	addz2 = -sqsz * x22;
	
	for (int i = 0; i < 1280 * 720; i++)
	{
		tmpx = i % 1280;
		tmpy = (i - tmpx) / 1280;

		vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
		vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
		vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

		vecltmp[i] = sqrt(vecn0*vecn0+vecn1*vecn1+vecn2*vecn2);
	}

	hipMemcpy(vecl, vecltmp, 1280 * 720 * sizeof(double), hipMemcpyHostToDevice);
}

void cudaExit()
{
	hipFree(buffer);
	hipFree(vecl);
	hipDeviceReset();
}

void cudathingy(uint8_t* pixels, bool flipg, double dc, double uhpy, double pos0, double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, double bhsize, double whsize)
{




	addKernel <<<(int)(1280 * 720 / 600), 600>>>(buffer, vecl, flipg, dc,uhpy, pos0,pos1,pos2,vec0,vec1,vec2,addy0,addy1,addy2,addz0,addz1,addz2,bhsize,whsize);


	hipDeviceSynchronize();

	hipMemcpy(pixels, buffer, 4 * 1280 * 720 * sizeof(uint8_t), hipMemcpyDeviceToHost);


}
